#include "hip/hip_runtime.h"
/*
 * SparseGridGpu single GPU benchmarks using the distributed interface.
 * 
 * Tommaso Bianucci, 17 Nov 2020
 */

#define BOOST_TEST_DYN_LINK
#define DISABLE_MPI_WRITTERS
// Generic perf-test stuff
#include <boost/test/unit_test.hpp>
// #include "performancePlots.hpp"
#include <iostream>
extern std::string suiteURI;
// extern report_sparse_grid_tests report_sparsegrid_funcs;q
// extern std::set<std::string> testSet;
// Specific include
#include "Grid/grid_dist_id.hpp"

// Work plan:
// - Insert benchmarks
// - Stencil benchmarks
// - Stencil insert benchmarks

template <typename SgridT, typename BoxT, typename cT>
float insertConcentricSpheres2D(SgridT &gdist, 
									BoxT box, 
									cT c, 
									const unsigned int pitch, 
									const float occupancy,
									const unsigned int iterations=1)
{
	// unsigned int r1 = 10, r2 = 10+r1;
	unsigned int r1 = (unsigned int) round(pitch*(1-occupancy)), r2 = pitch;

	typedef typename GetAddBlockType<SgridT>::type InsertBlockT;

	auto c0=c[0], c1=c[1];

	// Start timing
	timer ts;
	hipDeviceSynchronize();
	ts.start();

	for (unsigned int it=0; it<iterations; ++it)
	{
		gdist.addPoints(box.getKP1(),box.getKP2(),
				        [c0,c1,r1,r2] __device__ (int i, int j)
				        {
				        	// This controls if to insert in position (i,j)
				        	// The interval for insertion is [r1,r2) (half-closed)
				        	float r = sqrtf(
				        					(i-c0)*(i-c0)
				        					+ (j-c1)*(j-c1)
				        				  );
				        	r = fmodf(r, r2);
							return r>=r1;
				        },
				        [it] __device__ (InsertBlockT & data, int i, int j)
				        {
				        	data.template get<0>() = it*(i + j);
				        }
				        );

		gdist.template flush<smax_<0>>(flush_type::FLUSH_ON_DEVICE);
		// gdist.template ghost_get<0,1>(RUN_ON_DEVICE);
	}

	// Stop timing
	hipDeviceSynchronize();
	ts.stop();

	gdist.template deviceToHost<0>();

    return ts.getwct();
}
template <typename SgridT, typename BoxT, typename cT>
float insertConcentricSpheres3D(SgridT &gdist, 
									BoxT box, 
									cT c, 
									const unsigned int pitch, 
									const float occupancy,
									const unsigned int iterations=1)
{
	// unsigned int r1 = 10, r2 = 10+r1;
	unsigned int r1 = (unsigned int) round(pitch*(1-occupancy)), r2 = pitch;

	typedef typename GetAddBlockType<SgridT>::type InsertBlockT;

	auto c0=c[0], c1=c[1], c2=c[2];

	// Start timing
	timer ts;
	hipDeviceSynchronize();
	ts.start();

	for (unsigned int it=0; it<iterations; ++it)
	{
		gdist.addPoints(box.getKP1(),box.getKP2(),
				        [c0,c1,c2,r1,r2] __device__ (int i, int j, int k)
				        {
				        	// This controls if to insert in position (i,j,k)
				        	// The interval for insertion is [r1,r2) (half-closed)
				        	float r = sqrtf(
				        					(i-c0)*(i-c0)
				        					+ (j-c1)*(j-c1)
				        					+ (k-c2)*(k-c2)
				        				  );
				        	r = fmodf(r, r2);
							return r>=r1;
				        },
				        [it] __device__ (InsertBlockT & data, int i, int j, int k)
				        {
				        	data.template get<0>() = it*(i + j + k);
				        }
				        );

		gdist.template flush<smax_<0>>(flush_type::FLUSH_ON_DEVICE);
		// gdist.template ghost_get<0,1>(RUN_ON_DEVICE);
	}

	// Stop timing
	hipDeviceSynchronize();
	ts.stop();

	gdist.template deviceToHost<0>();

    return ts.getwct();
}

template <typename SgridT, typename BoxT>
float insertFullGrid2D(SgridT &gdist, 
						BoxT box,
						const unsigned int iterations=1)
{
	typedef typename GetAddBlockType<SgridT>::type InsertBlockT;

	// Start timing
	timer ts;
	hipDeviceSynchronize();
	ts.start();

	for (unsigned int it=0; it<iterations; ++it)
	{
		gdist.addPoints(box.getKP1(),box.getKP2(),
				        [] __device__ (int i, int j)
				        {
				        	// This controls if to insert in position (i,j,k)
							return true;
				        },
				        [it] __device__ (InsertBlockT & data, int i, int j)
				        {
				        	data.template get<0>() = it*(i + j);
				        }
				        );

		gdist.template flush<smax_<0>>(flush_type::FLUSH_ON_DEVICE);
		// gdist.template ghost_get<0,1>(RUN_ON_DEVICE);
	}

	// Stop timing
	hipDeviceSynchronize();
	ts.stop();

	gdist.template deviceToHost<0>();

    return ts.getwct();
}
template <typename SgridT, typename BoxT>
float insertFullGrid3D(SgridT &gdist, 
						BoxT box,
						const unsigned int iterations=1)
{
	typedef typename GetAddBlockType<SgridT>::type InsertBlockT;

	// Start timing
	timer ts;
	hipDeviceSynchronize();
	ts.start();

	for (unsigned int it=0; it<iterations; ++it)
	{
		gdist.addPoints(box.getKP1(),box.getKP2(),
				        [] __device__ (int i, int j, int k)
				        {
				        	// This controls if to insert in position (i,j,k)
							return true;
				        },
				        [it] __device__ (InsertBlockT & data, int i, int j, int k)
				        {
				        	data.template get<0>() = it*(i + j + k);
				        }
				        );

		gdist.template flush<smax_<0>>(flush_type::FLUSH_ON_DEVICE);
		// gdist.template ghost_get<0,1>(RUN_ON_DEVICE);
	}

	// Stop timing
	hipDeviceSynchronize();
	ts.stop();

	gdist.template deviceToHost<0>();

    return ts.getwct();
}

BOOST_AUTO_TEST_SUITE(performance)
BOOST_AUTO_TEST_SUITE(SparseGridGpu_dist_single)

BOOST_AUTO_TEST_SUITE(dim_2D)

BOOST_AUTO_TEST_CASE(insert_spheres)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;
	BOOST_TEST_CHECKPOINT("Starting...");
	size_t sz[2] = {10000,10000};
	periodicity<2> bc = {PERIODIC,PERIODIC};
	Ghost<2,long int> g(1);
	Box<2,float> domain({0.0,0.0},{1.0,1.0});
	sgrid_dist_id_gpu<2,float,aggregate<float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<2,size_t> box({1,1},{sz[0]-1,sz[1]-1});
	Box<2,size_t> box({0,0},{sz[0]-1,sz[1]-1});

	// Insert the concentric spheres on GPU
	const float occupancy = 0.5;
	const unsigned int pitch = 32;
	const unsigned int iterations = 1000;
    size_t c[3] = { sz[0]/2, sz[1]/2, 0 };

    auto elapsedTime = insertConcentricSpheres2D(gdist, box, c, pitch, occupancy, iterations);

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1];
    auto insertionRate = 1e-9*numElements*iterations/elapsedTime; // In MElem/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Insertion rate: " << insertionRate << " GElem/s" << std::endl;
	BOOST_TEST_CHECKPOINT("Ending...");
}

BOOST_AUTO_TEST_CASE(insert_full)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[2] = {10000,10000};
	periodicity<2> bc = {PERIODIC,PERIODIC};
	Ghost<2,long int> g(1);
	Box<2,float> domain({0.0,0.0},{1.0,1.0});
	sgrid_dist_id_gpu<2,float,aggregate<float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<2,size_t> box({1,1},{sz[0]-1,sz[1]-1});
	Box<2,size_t> box({0,0},{sz[0]-1,sz[1]-1});

	const unsigned int iterations = 1000;

	// Insert full on GPU
    auto elapsedTime = insertFullGrid2D(gdist, box, iterations);

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1];
    auto insertionRate = 1e-9*numElements*iterations/elapsedTime; // In MElem/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Insertion rate: " << insertionRate << " GElem/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}

BOOST_AUTO_TEST_CASE(stencil_spheres)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[2] = {10000,10000};
	periodicity<2> bc = {PERIODIC,PERIODIC};
	Ghost<2,long int> g(1);
	Box<2,float> domain({0.0,0.0},{1.0,1.0});
	sgrid_dist_id_gpu<2,float,aggregate<float, float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<2,size_t> box({1,1},{sz[0]-1,sz[1]-1});
	Box<2,size_t> box({0,0},{sz[0]-1,sz[1]-1});

	// Insert the concentric spheres on GPU
	const float occupancy = 0.1;
	const unsigned int pitch = 32;
	// const unsigned int pitch = 10;
	// const unsigned int pitch = 100;
    size_t c[3] = { sz[0]/2, sz[1]/2, 0};

    auto elapsedTime_insert = insertConcentricSpheres2D(gdist, box, c, pitch, occupancy);

    // Convolve a stencil
    // GetCpBlockType<GridType, property, stencilSize>
    // typedef typename GetCpBlockType<decltype(gdist),0,1>::type CpBlockType;

    const unsigned int iterations = 1000;

    timer ts;
    hipDeviceSynchronize();
    ts.start();

    for (unsigned int it=0; it<iterations; ++it)
    {
		gdist.template conv_cross<0,1,1>(
			{2,2,0},
			{(int)sz[0]-3,(int)sz[1]-3,0},
				[] __device__ (float & u, 
								cross_stencil<2,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						- 4.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
		gdist.template conv_cross<1,0,1>(
			{2,2,0},
			{(int)sz[0]-3,(int)sz[1]-3,0},
				[] __device__ (float & u, 
								cross_stencil<2,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						- 4.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
	}

	// hipDeviceSynchronize();
	ts.stop();

	gdist.deviceToHost<0,1>();

	float elapsedTime = ts.getwct();

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1];
    auto processingRate = 1e-9*numElements*2*iterations/elapsedTime; // In GElem/s
    auto gflops = 7*processingRate; // GFlops/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Processing rate: " << processingRate << " GElem/s" << std::endl;
	std::cout << ":: Throughput: " << gflops << " GFlops/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}
BOOST_AUTO_TEST_CASE(stencil_full)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[2] = {10000,10000};
	periodicity<2> bc = {PERIODIC,PERIODIC};
	Ghost<2,long int> g(1);
	Box<2,float> domain({0.0,0.0},{1.0,1.0});
	sgrid_dist_id_gpu<2,float,aggregate<float, float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<2,size_t> box({1,1},{sz[0]-1,sz[1]-1});
	Box<2,size_t> box({0,0},{sz[0]-1,sz[1]-1});

	// Insert full on GPU
    auto elapsedTime_insert = insertFullGrid2D(gdist, box);

    // Convolve a stencil
    // GetCpBlockType<GridType, property, stencilSize>
    // typedef typename GetCpBlockType<decltype(gdist),0,1>::type CpBlockType;

    const unsigned int iterations = 1000;

    timer ts;
    hipDeviceSynchronize();
    ts.start();

    for (unsigned int it=0; it<iterations; ++it)
    {
		gdist.template conv_cross<0,1,1>(
			{2,2,0},
			{(int)sz[0]-3,(int)sz[1]-3,0},
				[] __device__ (float & u, 
								cross_stencil<2,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						- 4.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
		gdist.template conv_cross<1,0,1>(
			{2,2,0},
			{(int)sz[0]-3,(int)sz[1]-3,0},
				[] __device__ (float & u, 
								cross_stencil<2,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						- 4.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
	}

	// hipDeviceSynchronize();
	ts.stop();

	gdist.deviceToHost<0,1>();

	float elapsedTime = ts.getwct();

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1];
    auto processingRate = 1e-9*numElements*2*iterations/elapsedTime; // In GElem/s
    auto gflops = 7*processingRate; // GFlops/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Processing rate: " << processingRate << " GElem/s" << std::endl;
	std::cout << ":: Throughput: " << gflops << " GFlops/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}

BOOST_AUTO_TEST_SUITE_END() //dim_2D

BOOST_AUTO_TEST_SUITE(dim_3D)

BOOST_AUTO_TEST_CASE(insert_spheres)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[3] = {2*500,2*500,500};
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	Ghost<3,long int> g(1);
	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});
	sgrid_dist_id_gpu<3,float,aggregate<float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<3,size_t> box({1,1,1},{sz[0]-1,sz[1]-1,sz[2]-1});
	Box<3,size_t> box({0,0,0},{sz[0]-1,sz[1]-1,sz[2]-1});

	// Insert the concentric spheres on GPU
	const float occupancy = 0.5;
	const unsigned int pitch = 32;
	const unsigned int iterations = 100;
    size_t c[3] = { sz[0]/2, sz[1]/2, sz[2]/2 };

    auto elapsedTime = insertConcentricSpheres3D(gdist, box, c, pitch, occupancy, iterations);

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1]*sz[2];
    auto insertionRate = 1e-9*numElements*iterations/elapsedTime; // In MElem/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Insertion rate: " << insertionRate << " GElem/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}

BOOST_AUTO_TEST_CASE(insert_full)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[3] = {2*500,2*500,500};
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	Ghost<3,long int> g(1);
	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});
	sgrid_dist_id_gpu<3,float,aggregate<float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<3,size_t> box({1,1,1},{sz[0]-1,sz[1]-1,sz[2]-1});
	Box<3,size_t> box({0,0,0},{sz[0]-1,sz[1]-1,sz[2]-1});

	const unsigned int iterations = 100;

	// Insert full on GPU
    auto elapsedTime = insertFullGrid3D(gdist, box, iterations);

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1]*sz[2];
    auto insertionRate = 1e-9*numElements*iterations/elapsedTime; // In MElem/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Insertion rate: " << insertionRate << " GElem/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}

BOOST_AUTO_TEST_CASE(stencil_spheres)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[3] = {2*500,2*500,500};
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	Ghost<3,long int> g(1);
	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});
	sgrid_dist_id_gpu<3,float,aggregate<float,float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<3,size_t> box({1,1,1},{sz[0]-1,sz[1]-1,sz[2]-1});
	Box<3,size_t> box({0,0,0},{sz[0]-1,sz[1]-1,sz[2]-1});

	// Insert the concentric spheres on GPU
	const float occupancy = 0.1;
	const unsigned int pitch = 32;
	// const unsigned int pitch = 10;
	// const unsigned int pitch = 100;
    size_t c[3] = { sz[0]/2, sz[1]/2, sz[2]/2 };

    auto elapsedTime_insert = insertConcentricSpheres3D(gdist, box, c, pitch, occupancy);

    // Convolve a stencil
    // GetCpBlockType<GridType, property, stencilSize>
    // typedef typename GetCpBlockType<decltype(gdist),0,1>::type CpBlockType;

    const unsigned int iterations = 500;

    timer ts;
    hipDeviceSynchronize();
    ts.start();

    for (unsigned int it=0; it<iterations; ++it)
    {
		gdist.template conv_cross<0,1,1>(
			{2,2,2},
			{(int)sz[0]-3,(int)sz[1]-3,(int)sz[2]-3},
				[] __device__ (float & u, 
								cross_stencil<3,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						+ cs.xm[2] + cs.xp[2] 
						- 6.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
		gdist.template conv_cross<1,0,1>(
			{2,2,2},
			{(int)sz[0]-3,(int)sz[1]-3,(int)sz[2]-3},
				[] __device__ (float & u, 
								cross_stencil<3,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						+ cs.xm[2] + cs.xp[2] 
						- 6.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
	}

	// hipDeviceSynchronize();
	ts.stop();

	gdist.deviceToHost<0,1>();

	float elapsedTime = ts.getwct();

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1]*sz[2];
    auto processingRate = 1e-9*numElements*2*iterations/elapsedTime; // In GElem/s
    auto gflops = 9*processingRate; // GFlops/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Processing rate: " << processingRate << " GElem/s" << std::endl;
	std::cout << ":: Throughput: " << gflops << " GFlops/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}
BOOST_AUTO_TEST_CASE(stencil_full)
{
	std::string fullName = boost::unit_test::framework::current_test_case().full_name();
	std::cout << ": Test: " << fullName << std::endl;

	size_t sz[3] = {2*500,2*500,500};
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	Ghost<3,long int> g(1);
	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});
	sgrid_dist_id_gpu<3,float,aggregate<float,float>> gdist(sz,domain,g,bc);
	gdist.template setBackgroundValue<0>(666);

	// Box<3,size_t> box({1,1,1},{sz[0]-1,sz[1]-1,sz[2]-1});
	Box<3,size_t> box({0,0,0},{sz[0]-1,sz[1]-1,sz[2]-1});

	// Insert full on GPU
    auto elapsedTime_insert = insertFullGrid3D(gdist, box);

    // Convolve a stencil
    // GetCpBlockType<GridType, property, stencilSize>
    // typedef typename GetCpBlockType<decltype(gdist),0,1>::type CpBlockType;

    const unsigned int iterations = 200;

    timer ts;
    hipDeviceSynchronize();
    ts.start();

    for (unsigned int it=0; it<iterations; ++it)
    {
		gdist.template conv_cross<0,1,1>(
			{2,2,2},
			{(int)sz[0]-3,(int)sz[1]-3,(int)sz[2]-3},
				[] __device__ (float & u, 
								cross_stencil<3,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						+ cs.xm[2] + cs.xp[2] 
						- 6.0*u)*0.1;
			}
		);
    	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
		gdist.template conv_cross<1,0,1>(
			{2,2,2},
			{(int)sz[0]-3,(int)sz[1]-3,(int)sz[2]-3},
				[] __device__ (float & u, 
								cross_stencil<3,float> & cs)
			{
				return u 
						+ (cs.xm[0] + cs.xp[0] 
						+ cs.xm[1] + cs.xp[1] 
						+ cs.xm[2] + cs.xp[2] 
						- 6.0*u)*0.1;
			}
		);
	hipDeviceSynchronize(); // We don't want an overlapping-kernel mess here!
	}

	// hipDeviceSynchronize();
	ts.stop();

	gdist.deviceToHost<0,1>();

	float elapsedTime = ts.getwct();

    auto numElements = gdist.size_local_inserted();
    size_t gridSize = sz[0]*sz[1]*sz[2];
    auto processingRate = 1e-9*numElements*2*iterations/elapsedTime; // In GElem/s
    auto gflops = 9*processingRate; // GFlops/s

	std::cout << ":: numElements: " << numElements << std::endl;
	std::cout << ":: gridSize: " << gridSize << std::endl;
	std::cout << ":: occupancy: " << ( (float) numElements ) / gridSize << std::endl;
	std::cout << ":: Time: " << elapsedTime << std::endl;
	std::cout << ":: Processing rate: " << processingRate << " GElem/s" << std::endl;
	std::cout << ":: Throughput: " << gflops << " GFlops/s" << std::endl;
	std::cout << std::endl; // Empty line at the end of test output
}

BOOST_AUTO_TEST_SUITE_END() //dim_3D
BOOST_AUTO_TEST_SUITE_END() //SparseGridGpu_dist_single
BOOST_AUTO_TEST_SUITE_END() //performance

//eof
